#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include <cuda/std/type_traits>
#include <cuda/std/limits>

#include "AC/Core/Image.hpp"

namespace ac::core::cuda
{
    template<typename Float, ::cuda::std::enable_if_t<::cuda::std::is_floating_point_v<Float>, bool> = true>
    __device__ inline Float fromFloat(const float v)
    {
        return fminf(fmaxf(v, 0.0f), 1.0f);
    }
    template<typename Unsigned, ::cuda::std::enable_if_t<::cuda::std::is_unsigned_v<Unsigned>, bool> = true>
    __device__ inline Unsigned fromFloat(const float v)
    {
        return static_cast<Unsigned>(fromFloat<float>(v) * ::cuda::std::numeric_limits<Unsigned>::max() + 0.5f);
    }

    __device__ inline static float dot(const float4 a, const float* const __restrict__ b)
    {
        return a.x * b[0] + a.y * b[1] + a.z * b[2] + a.w * b[3];
    }

    template<int cout,
        ::cuda::std::enable_if_t<cout % 4 == 0 && (cout * 9 <= 128 * 4), bool> = true>
    __global__ void conv3x3_cuda_cin1(
        hipTextureObject_t src,
        hipSurfaceObject_t dst,
        const unsigned int width,
        const unsigned int height,
        const float* const __restrict__ kernels,
        const float* const __restrict__ biases
    )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;
        auto tid = threadIdx.y * blockDim.x + threadIdx.x;
        
        __shared__ float kptr[cout * 9];
        if (tid * 4 < cout * 9)
        {
            kptr[tid * 4 + 0] = kernels[tid * 4 + 0];
            kptr[tid * 4 + 1] = kernels[tid * 4 + 1];
            kptr[tid * 4 + 2] = kernels[tid * 4 + 2];
            kptr[tid * 4 + 3] = kernels[tid * 4 + 3];
        }
        __syncthreads();

        if (x >= width || y >= height) return;

        constexpr int lout = cout / 4;

        const float r[] = {
            tex2D<float>(src, x - 1, y - 1),
            tex2D<float>(src, x    , y - 1),
            tex2D<float>(src, x + 1, y - 1),
            tex2D<float>(src, x - 1, y    ),
            tex2D<float>(src, x    , y    ),
            tex2D<float>(src, x + 1, y    ),
            tex2D<float>(src, x - 1, y + 1),
            tex2D<float>(src, x    , y + 1),
            tex2D<float>(src, x + 1, y + 1)
        };

        for (int nidx = 0; nidx < lout; nidx++)
        {
            auto npos = nidx * 4;

            auto offset0 = (npos + 0) * 9;
            auto offset1 = (npos + 1) * 9;
            auto offset2 = (npos + 2) * 9;
            auto offset3 = (npos + 3) * 9;

            auto layer = make_ushort4(
                __half_as_ushort(__float2half(fmaxf(
                    r[0] * kptr[offset0 + 0] +
                    r[1] * kptr[offset0 + 1] +
                    r[2] * kptr[offset0 + 2] +
                    r[3] * kptr[offset0 + 3] +
                    r[4] * kptr[offset0 + 4] +
                    r[5] * kptr[offset0 + 5] +
                    r[6] * kptr[offset0 + 6] +
                    r[7] * kptr[offset0 + 7] +
                    r[8] * kptr[offset0 + 8] + biases[npos + 0], 0.0f
                ))),
                __half_as_ushort(__float2half(fmaxf(
                    r[0] * kptr[offset1 + 0] +
                    r[1] * kptr[offset1 + 1] +
                    r[2] * kptr[offset1 + 2] +
                    r[3] * kptr[offset1 + 3] +
                    r[4] * kptr[offset1 + 4] +
                    r[5] * kptr[offset1 + 5] +
                    r[6] * kptr[offset1 + 6] +
                    r[7] * kptr[offset1 + 7] +
                    r[8] * kptr[offset1 + 8] + biases[npos + 1], 0.0f
                ))),
                __half_as_ushort(__float2half(fmaxf(
                    r[0] * kptr[offset2 + 0] +
                    r[1] * kptr[offset2 + 1] +
                    r[2] * kptr[offset2 + 2] +
                    r[3] * kptr[offset2 + 3] +
                    r[4] * kptr[offset2 + 4] +
                    r[5] * kptr[offset2 + 5] +
                    r[6] * kptr[offset2 + 6] +
                    r[7] * kptr[offset2 + 7] +
                    r[8] * kptr[offset2 + 8] + biases[npos + 2], 0.0f
                ))),
                __half_as_ushort(__float2half(fmaxf(
                    r[0] * kptr[offset3 + 0] +
                    r[1] * kptr[offset3 + 1] +
                    r[2] * kptr[offset3 + 2] +
                    r[3] * kptr[offset3 + 3] +
                    r[4] * kptr[offset3 + 4] +
                    r[5] * kptr[offset3 + 5] +
                    r[6] * kptr[offset3 + 6] +
                    r[7] * kptr[offset3 + 7] +
                    r[8] * kptr[offset3 + 8] + biases[npos + 3], 0.0f
                ))));
            surf2DLayeredwrite(layer, dst, sizeof(layer) * x, y, nidx, hipBoundaryModeZero);
        }
    }

    template<int cin, int cout, bool residual = false,
        ::cuda::std::enable_if_t<(cin % 4 == 0) && (cout % 4 == 0) && (cout * 9 * cin <= 128 * 8), bool> = true>
    __global__ void conv3x3_cuda(
        hipTextureObject_t src,
        hipSurfaceObject_t dst,
        const unsigned int width,
        const unsigned int height,
        const float* const __restrict__ kernels,
        const float* const __restrict__ biases
    )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;
        auto tid = threadIdx.y * blockDim.x + threadIdx.x;

        __shared__ float kptr[cout * 9 * cin];
        if (tid * 8 < cout * 9 * cin)
        {
            kptr[tid * 8 + 0] = kernels[tid * 8 + 0];
            kptr[tid * 8 + 1] = kernels[tid * 8 + 1];
            kptr[tid * 8 + 2] = kernels[tid * 8 + 2];
            kptr[tid * 8 + 3] = kernels[tid * 8 + 3];
            kptr[tid * 8 + 4] = kernels[tid * 8 + 4];
            kptr[tid * 8 + 5] = kernels[tid * 8 + 5];
            kptr[tid * 8 + 6] = kernels[tid * 8 + 6];
            kptr[tid * 8 + 7] = kernels[tid * 8 + 7];
        }
        __syncthreads();

        if (x >= width || y >= height) return;

        constexpr int lin = cin / 4;
        constexpr int lout = cout / 4;

        float4 r0[lin] = {};
        float4 r1[lin] = {};
        float4 r2[lin] = {};
        float4 r3[lin] = {};
        float4 r4[lin] = {};
        float4 r5[lin] = {};
        float4 r6[lin] = {};
        float4 r7[lin] = {};
        float4 r8[lin] = {};

        for (int cidx = 0; cidx < lin; cidx++)
        {
            r0[cidx] = tex2DLayered<float4>(src, x - 1, y - 1, cidx);
            r1[cidx] = tex2DLayered<float4>(src, x    , y - 1, cidx);
            r2[cidx] = tex2DLayered<float4>(src, x + 1, y - 1, cidx);
            r3[cidx] = tex2DLayered<float4>(src, x - 1, y    , cidx);
            r4[cidx] = tex2DLayered<float4>(src, x    , y    , cidx);
            r5[cidx] = tex2DLayered<float4>(src, x + 1, y    , cidx);
            r6[cidx] = tex2DLayered<float4>(src, x - 1, y + 1, cidx);
            r7[cidx] = tex2DLayered<float4>(src, x    , y + 1, cidx);
            r8[cidx] = tex2DLayered<float4>(src, x + 1, y + 1, cidx);
        };

        for (int nidx = 0; nidx < lout; nidx++)
        {
            auto npos = nidx * 4;
            float sum[4] = {};
            for (int i = 0; i < 4; i++)
            {
                auto offset0 = (npos + i) * 9 * cin + 0 * cin;
                auto offset1 = (npos + i) * 9 * cin + 1 * cin;
                auto offset2 = (npos + i) * 9 * cin + 2 * cin;
                auto offset3 = (npos + i) * 9 * cin + 3 * cin;
                auto offset4 = (npos + i) * 9 * cin + 4 * cin;
                auto offset5 = (npos + i) * 9 * cin + 5 * cin;
                auto offset6 = (npos + i) * 9 * cin + 6 * cin;
                auto offset7 = (npos + i) * 9 * cin + 7 * cin;
                auto offset8 = (npos + i) * 9 * cin + 8 * cin;

                for (int cidx = 0; cidx < lin; cidx++)
                {
                    auto cpos = cidx * 4;
                    sum[i] +=
                        dot(r0[cidx], kptr + offset0 + cpos) +
                        dot(r1[cidx], kptr + offset1 + cpos) +
                        dot(r2[cidx], kptr + offset2 + cpos) +
                        dot(r3[cidx], kptr + offset3 + cpos) +
                        dot(r4[cidx], kptr + offset4 + cpos) +
                        dot(r5[cidx], kptr + offset5 + cpos) +
                        dot(r6[cidx], kptr + offset6 + cpos) +
                        dot(r7[cidx], kptr + offset7 + cpos) +
                        dot(r8[cidx], kptr + offset8 + cpos);
                }

                sum[i] += biases[npos + i];
            }
            if constexpr (residual)
            {
                auto res = surf2DLayeredread<ushort4>(dst, sizeof(ushort4) * x, y, nidx, hipBoundaryModeZero);
                sum[0] += __half2float(__ushort_as_half(res.x));
                sum[1] += __half2float(__ushort_as_half(res.y));
                sum[2] += __half2float(__ushort_as_half(res.z));
                sum[3] += __half2float(__ushort_as_half(res.w));
            }
            auto layer = make_ushort4(
                __half_as_ushort(__float2half(fmaxf(sum[0], 0.0f))),
                __half_as_ushort(__float2half(fmaxf(sum[1], 0.0f))),
                __half_as_ushort(__float2half(fmaxf(sum[2], 0.0f))),
                __half_as_ushort(__float2half(fmaxf(sum[3], 0.0f))));

            surf2DLayeredwrite(layer, dst, sizeof(layer) * x, y, nidx, hipBoundaryModeZero);
        }
    }

    template<typename OUT, int cin,
        ::cuda::std::enable_if_t<cin % 4 == 0, bool> = true>
    __global__ void deconv2x2_cuda_cout1(
        hipTextureObject_t src,
        hipSurfaceObject_t dst,
        const unsigned int width,
        const unsigned int height,
        const float* const __restrict__ kernels
    )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width || y >= height) return;

        constexpr int lin = cin / 4;

        const unsigned int index = ((y & 1) << 1) + (x & 1);

        float sum = 0.0f;
        for (int cidx = 0; cidx < lin; cidx++) sum += dot(tex2DLayered<float4>(src, x / 2, y / 2, cidx), kernels + cin * index + cidx * 4);
        surf2Dwrite(fromFloat<OUT>(sum), dst, sizeof(OUT) * x, y, hipBoundaryModeZero);
    }

    void conv3x3_1to8_cuda(
        hipTextureObject_t src,
        hipSurfaceObject_t dst,
        unsigned int width,
        unsigned int height,
        const float* kernels,
        const float* biases,
        hipStream_t stream
    ) noexcept
    {
        dim3 block{ 16, 8 };
        dim3 grid{ (width + block.x - 1) / block.x, (height + block.y - 1) / block.y };
        conv3x3_cuda_cin1<8> <<< grid, block, 0, stream >>> (src, dst, width, height, kernels, biases);
    }

    void conv3x3_8to8_cuda(
        hipTextureObject_t src,
        hipSurfaceObject_t dst,
        unsigned int width,
        unsigned int height,
        const float* kernels,
        const float* biases,
        hipStream_t stream
    ) noexcept
    {
        dim3 block{ 16, 8 };
        dim3 grid{ (width + block.x - 1) / block.x, (height + block.y - 1) / block.y };
        conv3x3_cuda<8, 8> <<< grid, block, 0, stream >>> (src, dst, width, height, kernels, biases);
    }

    void conv3x3_residual_8to8_cuda(
        hipTextureObject_t src,
        hipSurfaceObject_t dst,
        unsigned int width,
        unsigned int height,
        const float* kernels,
        const float* biases,
        hipStream_t stream
    ) noexcept
    {
        dim3 block{ 16, 8 };
        dim3 grid{ (width + block.x - 1) / block.x, (height + block.y - 1) / block.y };
        conv3x3_cuda<8, 8, true> <<< grid, block, 0, stream >>> (src, dst, width, height, kernels, biases);
    }

    void deconv2x2_8to1_cuda(
        hipTextureObject_t src,
        hipSurfaceObject_t dst,
        unsigned int width,
        unsigned int height,
        const float* kernels,
        Image::ElementType type,
        hipStream_t stream
    ) noexcept
    {
        dim3 block{ 16, 8 };
        dim3 grid{ (width + block.x - 1) / block.x, (height + block.y - 1) / block.y };
        switch (type)
        {
        case Image::UInt8:
            return deconv2x2_cuda_cout1<std::uint8_t, 8> <<< grid, block, 0, stream >>> (src, dst, width, height, kernels);
        case Image::UInt16:
            return deconv2x2_cuda_cout1<std::uint16_t, 8> <<< grid, block, 0, stream >>> (src, dst, width, height, kernels);
        case Image::Float32:
            return deconv2x2_cuda_cout1<float, 8> <<< grid, block, 0, stream >>> (src, dst, width, height, kernels);
        }
    }
}
